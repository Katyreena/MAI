#include "hip/hip_runtime.h"
#define __HIPCC__
#include <hip/device_functions.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

#define THREADS = 256;
#define BLOCKS = 64;

__global__ void StandartDeviation(float* numbers, float* result, int n, float x)
{
	__shared__ float cache[THREADS];
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int cacheId = threadIdx.x;
	int totalNumberOfThreads = gridDim.x * blockDim.x;
	float tmp = 0;
	while (idx < n)
	{
		tmp += (sample[idx] - x) * (sample[idx] - x);
		idx += totalNumberOfThreads;
	}

	cache[cacheId] = tmp;
	__syncthreads();
	int iter = blockDim.x / 2;
	while (iter != 0)
	{
		if (cacheId < iter) cache[cacheId] += cache[cacheId + iter];
		__syncthreads();
		iter /= 2;
	}

	if (cacheId == 0) result[blockIdx.x] = cache[0];
}

void main()
{
	int N;
	printf("Print count of sample: \n");
	while (scanf("%d", &N) != 1)
	{
		printf("Incorrect! Try again: ");
		while (getchar() != '\n');
	}

	float x = 0;
	float* a, *b, *c_result;
	float *dev_sample, *dd, *dev_result;
	a = (float*)malloc(N * sizeof(float));
	b = (float*)malloc(N * sizeof(float));
	c_result = (float*)malloc(BLOCKS * sizeof(float));
	if (hipMalloc(&dev_sample, N * sizeof(float)) != hipSuccess) printf("Error: hipMalloc");
	if (hipMalloc(&dd, N * sizeof(float)) != hipSuccess) printf("Error: hipMalloc");
	if (hipMalloc(&dev_result, BLOCKS * sizeof(float)) != hipSuccess) printf("Error: hipMalloc");
	printf("Print %d numbers: \n", N);
	for (int i = 0; i != N; i++)
	{
		float tmp = 0;
		while ((scanf("%f", &tmp)) != 1)
		{
			printf("Incorrect! Try again: ");
			while (getchar() != '\n');
		}
		x += tmp;
		a[i] = tmp;
	}

	if (hipMemcpy(dev_sample, a, N * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) printf("Error: hipMemcpy!");
	if (hipMemcpy(dd, b, N * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) printf("Error: hipMemcpy!");
	StandartDeviation<<<BLOCKS, THREADS>>>(dev_sample, dev_result, N, (float)x/n);
	if (hipGetLastError() != hipSuccess) printf("Error: sumOfSquares");
	if (hipMemcpy(partial_c, dev_partial_c, sizeof(float) * blocksPerGrid, hipMemcpyDeviceToHost) != hipSuccess) printf("Error: hipMemcpy");
	if (hipGetLastError() != hipSuccess) printf("Error: sumOfSquares");
	if (hipMemcpy(c_result, dev_result, sizeof(float) * BLOCKS, hipMemcpyDeviceToHost) != hipSuccess) printf("Error: hipMemcpy");
	float c = 0;
	for (int i = 0; i != BLOCKS; i++) c += c_result[i];
	printf("Result: %f", c);
}
